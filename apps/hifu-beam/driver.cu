#include "hip/hip_runtime.h"
//=======================================================================
//
// Name : Finite Volume Nonlinear Acoustics GPU Implementation (FiVoNAGI)
//
// Authors : Roberto Velasco Segura and Pablo L. Rend\'on
//
// License : see licence.txt in the root directory of the repository.
//
//=======================================================================

// For comparison with CLAWPACK see: clawpack-4.6.1/apps/acoustics/2d/example1/driver.f
// and clawpack-4.6.1/apps/acoustics/2d/example1/Makefile

#define EXPAND1(x) x
#define EXPAND2(x) #x
#define CONCAT_STR(x, y) EXPAND1(x)y
#define QUOTE(x) EXPAND2(x)

#ifndef ROOT
# error ROOT variable not set.
#else
#define ROOT_PATH(FILE) QUOTE(CONCAT_STR(EXPAND1(ROOT), EXPAND1(FILE)))
#endif /* ROOT */

// use 1 for single precision and 2 for double precision
#ifndef PRECISION
#define PRECISION 1
#endif /* PRECISION */

#if PRECISION == 1 
#define DATATYPEV float
#define DATATYPET float
#elif PRECISION == 2
#define DATATYPEV double
#define DATATYPET int2
#else /* PRECISION value */
# error unresolved PRECISION value
#endif /* PRECISION value */

#include ROOT_PATH(/common/numbers.h)
#include "parameters.h"
#include ROOT_PATH(/common/data_definitions.h)
#include "init.h"
#include "boundary.h"
#include ROOT_PATH(/common/filter.h)
#include "draw_float_cut.h"
#include "data_export.h"
#include "data_collect.h"
#include "../na/source.h"
#include "../na/fv.h"

#include ROOT_PATH(/common/main.cu)
